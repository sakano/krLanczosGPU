#include "hip/hip_runtime.h"
﻿#include "lanczosGPU.hpp"

#include <vector>
#include <exception>
#define M_PI 3.14159265358979323846

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define checkCudaError(statement) \
    {\
        hipError_t error = statement; \
        if (error != hipSuccess) { \
            const char *mes = hipGetErrorString(error); \
            printf("%s", mes); \
            throw std::runtime_error(mes); \
        } \
    }
namespace
{
    using tjs_uint32 = GPU::tjs_uint32;
    
    template<int TTap>
    struct LanczosWeight
    {
        double operator()(double phase)
        {
            if (std::abs(phase) < DBL_EPSILON) return 1.0;
            if (std::abs(phase) >= (double)TTap) return 0.0;
            return std::sin(M_PI*phase)*std::sin(M_PI*phase / TTap) / (M_PI*M_PI*phase*phase / TTap);
        }
    };

    struct AxisParam
    {
        std::vector<int> start_;	// 開始インデックス
        std::vector<int> length_;	// 各要素長さ
        std::vector<double> weight_;
        std::vector<int> index_; // weight開始インデックス

        template<typename TWeightFunc>
        void calculateAxis(int srcstart, int srcend, int srclength, int dstlength, double tap, TWeightFunc& func);
    };

    // srclength = srcwidth
    // dstlength = dstwidth
    // srcstart = srcleft;
    // srcend = srcright;
    template<typename TWeightFunc>
    void AxisParam::calculateAxis(int srcstart, int srcend, int srclength, int dstlength, double tap, TWeightFunc& func)
    {
        start_.clear();
        start_.reserve(dstlength);
        length_.clear();
        length_.reserve(dstlength);
        index_.clear();
        index_.reserve(dstlength);
        int index = 0;
        if (srclength <= dstlength) { // 拡大
            double rangex = tap;
            int length = dstlength * (int)rangex * 2 + dstlength;
            weight_.reserve(length);
            for (int x = 0; x < dstlength; x++) {
                double cx = (x + 0.5)*(double)srclength / (double)dstlength + srcstart;
                int left = (int)std::floor(cx - rangex);
                int right = (int)std::floor(cx + rangex);
                if (left < srcstart) left = srcstart;
                if (right >= srcend) right = srcend;
                start_.push_back(left);
                int len = 0;
                for (int sx = left; sx < right; sx++) {
                    double dist = std::abs(sx + 0.5 - cx);
                    double weight = func(dist);
                    len++;
                    weight_.push_back(weight);
                }
                length_.push_back(len);
                index_.push_back(index);
                index += len;
            }
        }
        else { // 縮小
            double rangex = tap*(double)srclength / (double)dstlength;
            int length = srclength * (int)rangex * 2 + srclength;
            weight_.reserve(length);
            for (int x = 0; x < dstlength; x++) {
                double cx = (x + 0.5)*(double)srclength / (double)dstlength + srcstart;
                int left = (int)std::floor(cx - rangex);
                int right = (int)std::floor(cx + rangex);
                if (left < srcstart) left = srcstart;
                if (right >= srcend) right = srcend;
                start_.push_back(left);
                // 転送先座標での位置
                double delta = (double)dstlength / (double)srclength;
                double dx = (left + 0.5) * delta;
                int len = 0;
                for (int sx = left; sx < right; sx++) {
                    double dist = std::abs(dx - (x + 0.5));
                    double weight = func(dist);
                    dx += delta;
                    len++;
                    weight_.push_back(weight);
                }
                length_.push_back(len);
                index_.push_back(index);
                index += len;
            }
        }
    }

    __global__ void kernel_weightCopy(
        tjs_uint32 * const d_destbuf,
        const unsigned int destleft, const unsigned int desttop, const unsigned int destwidth, const unsigned int destheight,
        const unsigned int srcwidth, const tjs_uint32 * __restrict__  const d_srcbuf,
        const int * const d_startX, const int * const d_lengthX, const double * const d_weightX, const int * const d_indexX,
        const int * const d_startY, const int * const d_lengthY, const double * const d_weightY, const int * const d_indexY
        )
    {
        const int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = threadIndex / destwidth;
        const int x = threadIndex - y * destwidth;
        if (y > destheight || x > destwidth) return;

        int wiy = d_indexY[y];
        const int top = d_startY[y];
        const int bottom = d_startY[y] + d_lengthY[y];
        const int left = d_startX[x];
        const int right = d_startX[x] + d_lengthX[x];
        double color_element[4] = { 0.0, 0.0, 0.0, 0.0 };
        double w_total = 0.0;
        for (int sy = top; sy < bottom; sy++) {
            int wix = d_indexX[x];
            for (int sx = left; sx < right; sx++) {
                const double weight = (d_weightX[wix]) * (d_weightY[wiy]);
                const tjs_uint32 color = d_srcbuf[sy * srcwidth + sx];
                color_element[0] += (color & 0xff) * weight;
                color_element[1] += ((color >> 8) & 0xff) * weight;
                color_element[2] += ((color >> 16) & 0xff) * weight;
                color_element[3] += ((color >> 24) & 0xff) * weight;
                ++wix;
                w_total += weight;
            }
            wiy++;
        }
        if (w_total != 0) {
            const double mul = 1.0 / w_total;
            color_element[0] *= mul;
            color_element[1] *= mul;
            color_element[2] *= mul;
            color_element[3] *= mul;
        }
        tjs_uint32 color = (tjs_uint32)((color_element[0] > 255) ? 255 : (color_element[0] < 0) ? 0 : color_element[0]);
        color += (tjs_uint32)((color_element[1] > 255) ? 255 : (color_element[1] < 0) ? 0 : color_element[1]) << 8;
        color += (tjs_uint32)((color_element[2] > 255) ? 255 : (color_element[2] < 0) ? 0 : color_element[2]) << 16;
        color += (tjs_uint32)((color_element[3] > 255) ? 255 : (color_element[3] < 0) ? 0 : color_element[3]) << 24;

        d_destbuf[y * destwidth + x] = color;
    }

    template<class T>
    struct DeviceBuffer
    {
        T *ptr;

        explicit DeviceBuffer(const unsigned int size) {
            checkCudaError(hipMalloc(&ptr, sizeof(T) * size));
        }

        ~DeviceBuffer() {
            checkCudaError(hipFree(ptr));
        }
    };
        
    struct DeviceAxisParam
    {
        int *start;
        int *length;
        double *weight;
        int *index;

        explicit DeviceAxisParam(const AxisParam&& param) {
            checkCudaError(hipMalloc(&start, sizeof(int) * param.start_.size()));
            checkCudaError(hipMalloc(&length, sizeof(int) * param.length_.size()));
            checkCudaError(hipMalloc(&weight, sizeof(double) * param.weight_.size()));
            checkCudaError(hipMalloc(&index, sizeof(int) * param.index_.size()));
            
            checkCudaError(hipMemcpy(start, param.start_.data(), sizeof(int) * param.start_.size(), hipMemcpyHostToDevice));
            checkCudaError(hipMemcpy(length, param.length_.data(), sizeof(int) * param.length_.size(), hipMemcpyHostToDevice));
            checkCudaError(hipMemcpy(weight, param.weight_.data(), sizeof(double) * param.weight_.size(), hipMemcpyHostToDevice));
            checkCudaError(hipMemcpy(index, param.index_.data(), sizeof(int) * param.index_.size(), hipMemcpyHostToDevice));
        }

        ~DeviceAxisParam() {
            checkCudaError(hipFree(start));
            checkCudaError(hipFree(length));
            checkCudaError(hipFree(weight));
            checkCudaError(hipFree(index));
        }
    };
}

template<int W>
void GPU::TVPLanczos(
    const int destpitch, tjs_uint32 * const destbuf,
    const unsigned int destleft, const unsigned int desttop, const unsigned int destwidth, const unsigned int destheight,
    const int srcpitch, const tjs_uint32 * const srcbuf,
    const unsigned int srcleft, const unsigned int srctop, const unsigned int srcwidth, const unsigned int srcheight) {

    // パラメータ導出
    LanczosWeight<W> weightfunc;
    AxisParam paramx, paramy;
    paramx.calculateAxis(srcleft, srcleft + srcwidth, srcwidth, destwidth, static_cast<double>(W), weightfunc);
    paramy.calculateAxis(srctop, srctop + srcheight, srcheight, destheight, static_cast<double>(W), weightfunc);

    // CPUからGPUへデータ転送
    DeviceAxisParam deviceParamX(std::move(paramx));
    DeviceAxisParam deviceParamY(std::move(paramy));

    DeviceBuffer<tjs_uint32> deviceDestBuf(destwidth * destheight);
    DeviceBuffer<tjs_uint32> deviceSrcBuf(srcwidth * srcheight);
    for (unsigned int y = srctop; y < srcheight; ++y) {
        checkCudaError(hipMemcpy(deviceSrcBuf.ptr + (y - srctop) * srcwidth, srcbuf + srcleft + y * srcpitch / 4, sizeof(tjs_uint32) * srcwidth, hipMemcpyHostToDevice));
    }

    // GPU設定検出
    int max_threads;
    if (hipDeviceGetAttribute(&max_threads, hipDeviceAttributeMaxThreadsPerBlock, 0) != hipSuccess) {
        throw std::runtime_error("hipDeviceGetAttribute failed.");
    }
    
    // フィルタ処理実行
    const int threadNum = max_threads;
    const int blockNum = (destwidth * destheight + threadNum - 1) / threadNum;
    kernel_weightCopy<<<blockNum, threadNum>>>(
        deviceDestBuf.ptr,
        destleft, desttop, destwidth, destheight,
        srcwidth, deviceSrcBuf.ptr,
        deviceParamX.start, deviceParamX.length, deviceParamX.weight, deviceParamX.index,
        deviceParamY.start, deviceParamY.length, deviceParamY.weight, deviceParamY.index);
    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());

    // GPUからCPUへ結果を転送
    for (unsigned int y = desttop; y < destheight; ++y) {
        checkCudaError(hipMemcpy(destbuf + destleft + y * destpitch / 4, deviceDestBuf.ptr + (y - desttop) * destwidth, sizeof(tjs_uint32) * destwidth, hipMemcpyDeviceToHost));
    }
}

template void GPU::TVPLanczos<2>(
    const int destpitch, tjs_uint32 * const destbuf,
    const unsigned int destleft, const unsigned int desttop, const unsigned int destwidth, const unsigned int destheight,
    const int srcpitch, const tjs_uint32 * const srcbuf,
    const unsigned int srcleft, const unsigned int srctop, const unsigned int srcwidth, const unsigned int srcheight);
template void GPU::TVPLanczos<3>(
    const int destpitch, tjs_uint32 * const destbuf,
    const unsigned int destleft, const unsigned int desttop, const unsigned int destwidth, const unsigned int destheight,
    const int srcpitch, const tjs_uint32 * const srcbuf,
    const unsigned int srcleft, const unsigned int srctop, const unsigned int srcwidth, const unsigned int srcheight);
